#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "CUDACoordinationkernel.hpp"
#include <iostream>

__global__ void getCoord(double *coordinates, double *coordination,
                         unsigned Nat, double Rsqr) {
  const int i = threadIdx.x + blockIdx.x; // * blockDim.x;
  double x = coordinates[3 * i];
  double y = coordinates[3 * i + 1];
  double z = coordinates[3 * i + 2];
  coordination[i] = 0.0;
  double dx, dy, dz;
  for (unsigned j = 0; j < Nat; j++) {
    if (i == j) {
      continue;
    }
    dx = x - coordinates[3 * j];
    dy = y - coordinates[3 * j + 1];
    dz = z - coordinates[3 * j + 2];
    if ((dx * dx + dy * dy + dz * dz) < Rsqr) {
      coordination[i] += 1.0;
    }
  }
}

__global__ void reduction(double *input, int nat) {
  const int tid = threadIdx.x;
  if (tid > nat) {
    input[tid] = 0;
  }

  auto step_size = 1;
  int number_of_threads = blockDim.x;

  while (number_of_threads > 0) {
    if (tid < number_of_threads) // still alive?
    {
      const auto fst = tid * step_size * 2;
      const auto snd = fst + step_size;
      input[fst] += input[snd];
    }

    step_size <<= 1;
    number_of_threads >>= 1;
  }
}

double getCoordination(std::vector<PLMD::Vector> positions, double R_0) {
  auto nat = positions.size();
  size_t nexpw2 = pow(2, ceil(log2(nat)));
  double *d_data;
  double *ncoords;
  hipMalloc(&d_data, 3 * nat * sizeof(double));
  hipMalloc(&ncoords, nexpw2 * sizeof(double));
  hipMemcpy(d_data , &positions[0][0], 3 *nat* sizeof(double),
               hipMemcpyHostToDevice);
  /*
  for (size_t i = 0; i < nat; i++) {
    hipMemcpy(d_data + i * 3, &positions[i][0], 3 * sizeof(double),
               hipMemcpyHostToDevice);
  }*/

  double Rsqr = R_0 * R_0;
  getCoord<<<nat, 1>>>(d_data, ncoords, nat, Rsqr);
  reduction<<<1, nexpw2 / 2>>>(ncoords, nat);
  double result;
  hipMemcpy(&result, ncoords, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_data);
  hipFree(ncoords);
  return result;
}
